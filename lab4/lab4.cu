#include "hip/hip_runtime.h"
#include <wb.h>
#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "CUDA error: ", hipGetErrorString(err));              \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
    }                                                                     \
  } while (0)
//@@ Define any useful program-wide constants here
#define TILE_WIDTH 4
#define MASK_WIDTH 3
//@@ Define constant memory for device kernel here
__constant__ float Mc[MASK_WIDTH][MASK_WIDTH][MASK_WIDTH];
__global__ void conv3d(float *input, float *output, const int z_size,
                       const int y_size, const int x_size) {
  //@@ Insert kernel code here
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;
  int row_o = blockIdx.y * TILE_WIDTH + ty;
  int col_o = blockIdx.x * TILE_WIDTH + tx;
  int hei_o = blockIdx.z * TILE_WIDTH + tz;
  int row_i = row_o - 1;
  int col_i = col_o - 1;
  int hei_i = hei_o - 1;
  float p = 0;
  __shared__ float tile[TILE_WIDTH + MASK_WIDTH - 1][TILE_WIDTH + MASK_WIDTH - 1][TILE_WIDTH + MASK_WIDTH - 1];
  if (row_i >= 0 && row_i < y_size && col_i >= 0 && col_i < x_size && hei_i >= 0 && hei_i < z_size) {
    tile[tz][ty][tx] = input[hei_i * x_size * y_size + row_i * x_size + col_i];
  } else {
    tile[tz][ty][tx] = 0;
  }
  __syncthreads();
  if (tx < TILE_WIDTH && ty < TILE_WIDTH && tz < TILE_WIDTH) {
    for (int i = 0; i < MASK_WIDTH; i++) {
      for (int j = 0; j < MASK_WIDTH; j++) {
        for (int k = 0; k < MASK_WIDTH; k++) {
          p += Mc[i][j][k] * tile[tz + i][ty + j][tx + k];
        }
      }
    }
    if (row_o < y_size && col_o < x_size && hei_o < z_size) {
      output[hei_o * x_size * y_size + row_o * x_size + col_o] = p;
    }
  }
}
int main(int argc, char *argv[]) {
  wbArg_t args;
  int z_size;
  int y_size;
  int x_size;
  int inputLength, kernelLength;
  float *hostInput;
  float *hostKernel;
  float *hostOutput;
  //@@ Initial deviceInput and deviceOutput here.
  float *deviceInput, *deviceOutput;
  args = wbArg_read(argc, argv);
  // Import data
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostKernel =
      (float *)wbImport(wbArg_getInputFile(args, 1), &kernelLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));
  // First three elements are the input dimensions
  z_size = hostInput[0];
  y_size = hostInput[1];
  x_size = hostInput[2];
  wbLog(TRACE, "The input size is ", z_size, "x", y_size, "x", x_size);
  assert(z_size * y_size * x_size == inputLength - 3);
  assert(kernelLength == 27);
  //@@ Allocate GPU memory here
  // Recall that inputLength is 3 elements longer than the input data
  // because the first  three elements were the dimensions
  hipMalloc((void **) &deviceInput, (inputLength - 3) * sizeof(float));
  hipMalloc((void **) &deviceOutput, (inputLength - 3) * sizeof(float));
  //@@ Copy input and kernel to GPU here
  // Recall that the first three elements of hostInput are dimensions and
  // do
  // not need to be copied to the gpu
  hipMemcpy(deviceInput, hostInput + 3, (inputLength - 3) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(Mc), hostKernel, kernelLength * sizeof(float));
  //@@ Initialize grid and block dimensions here
  dim3 DimGrid(ceil((float) x_size / TILE_WIDTH), ceil((float) y_size / TILE_WIDTH), ceil((float) z_size / TILE_WIDTH));
  dim3 DimBlock(TILE_WIDTH + MASK_WIDTH - 1, TILE_WIDTH + MASK_WIDTH - 1, TILE_WIDTH + MASK_WIDTH - 1);
  //@@ Launch the GPU kernel here
  conv3d<<<DimGrid, DimBlock>>>(deviceInput, deviceOutput, z_size, y_size, x_size);
  hipDeviceSynchronize();
  //@@ Copy the device memory back to the host here
  // Recall that the first three elements of the output are the dimensions
  // and should not be set here (they are set below)
  hipMemcpy(hostOutput + 3, deviceOutput, (inputLength - 3) * sizeof(float), hipMemcpyDeviceToHost);
  // Set the output dimensions for correctness checking
  hostOutput[0] = z_size;
  hostOutput[1] = y_size;
  hostOutput[2] = x_size;
  wbSolution(args, hostOutput, inputLength);
  //@@ Free device memory
  hipFree(deviceInput);
  hipFree(deviceOutput);
  // Free host memory
  free(hostInput);
  free(hostOutput);
  return 0;
}