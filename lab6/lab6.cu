#include "hip/hip_runtime.h"
// MP Scan
// Given a list (lst) of length n
// Output its prefix sum = {lst[0], lst[0] + lst[1], lst[0] + lst[1] + ...
// +
// lst[n-1]}
#include <wb.h>
#define BLOCK_SIZE 512 //@@ You can change this
#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)
__global__ void scan(float *input, float *output, int len) {
  //@@ Modify the body of this function to complete the functionality of
  //@@ the scan on the device
  //@@ You may need multiple kernel calls; write your kernels before this
  //@@ function and call them from the host
  __shared__ float T[2 * BLOCK_SIZE];
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    T[threadIdx.x] = input[i];
  }
  int stride = 1;
  while (stride < 2 * BLOCK_SIZE) {
    __syncthreads();
    int index = (threadIdx.x + 1) * stride * 2 - 1;
    if (index < 2 * BLOCK_SIZE && index >= stride) {
      T[index] += T[index - stride];
    }
    stride *= 2;
  }
  stride = BLOCK_SIZE / 2;
  while (stride > 0) {
    __syncthreads();
    int index = (threadIdx.x + 1) * stride * 2 - 1;
    if (index + stride < 2 * BLOCK_SIZE) {
      T[index + stride] += T[index];
    }
    stride /= 2;
  }
  __syncthreads();
  if (i < len && i < BLOCK_SIZE) {
    output[i] = T[threadIdx.x];
  }
}
int main(int argc, char **argv) {
  wbArg_t args;
  float *hostInput;  // The input 1D list
  float *hostOutput; // The output list
  float *deviceInput;
  float *deviceOutput;
  int numElements; // number of elements in the list
  args = wbArg_read(argc, argv);
  // Import data and create memory on host
  // The number of input elements in the input is numElements
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &numElements);
  hostOutput = (float *)malloc(numElements * sizeof(float));
  // Allocate GPU memory.
  wbCheck(hipMalloc((void **)&deviceInput, numElements * sizeof(float)));
  wbCheck(hipMalloc((void **)&deviceOutput, numElements * sizeof(float)));
  // Clear output memory.
  wbCheck(hipMemset(deviceOutput, 0, numElements * sizeof(float)));
  // Copy input memory to the GPU.
  wbCheck(hipMemcpy(deviceInput, hostInput, numElements * sizeof(float),
                     hipMemcpyHostToDevice));
  //@@ Initialize the grid and block dimensions here
  dim3 DimGrid(ceil((float) numElements / BLOCK_SIZE), 1, 1);
  dim3 DimBlock(BLOCK_SIZE, 1, 1);
  //@@ Modify this to complete the functionality of the scan
  //@@ on the deivce
  scan<<<DimGrid, DimBlock>>>(deviceInput, deviceOutput, numElements);
  for (unsigned int i = 1; i < DimGrid.x; i++) {
    float temp;
    hipMemcpy(&temp, &deviceOutput[i * BLOCK_SIZE - 1], sizeof(float), hipMemcpyDeviceToHost);
    temp += hostInput[i * BLOCK_SIZE];
    hipMemcpy(&deviceInput[i * BLOCK_SIZE], &temp, sizeof(float), hipMemcpyHostToDevice);
    scan<<<DimGrid, DimBlock>>>(&deviceInput[i * BLOCK_SIZE], &deviceOutput[i * BLOCK_SIZE], numElements);
  }
  hipDeviceSynchronize();
  // Copying output memory to the CPU
  wbCheck(hipMemcpy(hostOutput, deviceOutput, numElements * sizeof(float),
                     hipMemcpyDeviceToHost));
  //@@  Free GPU Memory
  hipFree(deviceInput);
  hipFree(deviceOutput);
  wbSolution(args, hostOutput, numElements);
  free(hostInput);
  free(hostOutput);
  return 0;
}