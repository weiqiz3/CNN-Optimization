#include "hip/hip_runtime.h"
// MP Scan
// Given a list (lst) of length n
// Output its prefix sum = {lst[0], lst[0] + lst[1], lst[0] + lst[1] + ...
// +
// lst[n-1]}

#include <wb.h>

#define BLOCK_SIZE 512 //@@ You can change this

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

__global__ void scan(float *input, float *output, int len) {
  //@@ Modify the body of this function to complete the functionality of
  //@@ the scan on the device
  //@@ You may need multiple kernel calls; write your kernels before this
  //@@ function and call them from the host
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostInput;  // The input 1D list
  float *hostOutput; // The output list
  float *deviceInput;
  float *deviceOutput;
  int numElements; // number of elements in the list

  args = wbArg_read(argc, argv);

  // Import data and create memory on host
  // The number of input elements in the input is numElements
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &numElements);
  hostOutput = (float *)malloc(numElements * sizeof(float));


  // Allocate GPU memory.
  wbCheck(hipMalloc((void **)&deviceInput, numElements * sizeof(float)));
  wbCheck(hipMalloc((void **)&deviceOutput, numElements * sizeof(float)));


  // Clear output memory.
  wbCheck(hipMemset(deviceOutput, 0, numElements * sizeof(float)));

  // Copy input memory to the GPU.
  wbCheck(hipMemcpy(deviceInput, hostInput, numElements * sizeof(float),
                     hipMemcpyHostToDevice));

  //@@ Initialize the grid and block dimensions here


  //@@ Modify this to complete the functionality of the scan
  //@@ on the deivce

  hipDeviceSynchronize();

  // Copying output memory to the CPU
  wbCheck(hipMemcpy(hostOutput, deviceOutput, numElements * sizeof(float),
                     hipMemcpyDeviceToHost));


  //@@  Free GPU Memory


  wbSolution(args, hostOutput, numElements);

  free(hostInput);
  free(hostOutput);

  return 0;
}

