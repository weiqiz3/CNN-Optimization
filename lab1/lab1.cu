#include "hip/hip_runtime.h"
// LAB 1
#include <wb.h>

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
  //@@ Insert code to implement vector addition here
}

int main(int argc, char **argv) {
  wbArg_t args;
  int inputLength;
  float *hostInput1;
  float *hostInput2;
  float *hostOutput;

  args = wbArg_read(argc, argv);
  //@@ Importing data and creating memory on host
  hostInput1 =
      (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostInput2 =
      (float *)wbImport(wbArg_getInputFile(args, 1), &inputLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));

  wbLog(TRACE, "The input length is ", inputLength);

  //@@ Allocate GPU memory here


  //@@ Copy memory to the GPU here


  //@@ Initialize the grid and block dimensions here


  //@@ Launch the GPU Kernel here to perform CUDA computation

  hipDeviceSynchronize();
  //@@ Copy the GPU memory back to the CPU here


  //@@ Free the GPU memory here


  wbSolution(args, hostOutput, inputLength);

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);

  return 0;
}
