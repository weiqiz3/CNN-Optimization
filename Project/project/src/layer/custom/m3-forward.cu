#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"
#include <mma.h>
using namespace nvcuda;
#define TILE_WIDTH 16
#define NUM_UNROLL 4
__global__ void matmul_conv_fused(const float * __restrict__ mask, const float * __restrict__ input, float * __restrict__ output,
                                  int Batch, int Map_out, int Channel, int Height, int Width, int K)
{
    /*
    TODO: Modify this function to implement the fused unroll-matmul-permute kernel.
    Function parameter definitions:
    mask - convolution kernel
    input - input
    output - output
    Batch - batch_size (number of images in x)
    Map_out - number of output feature maps
    Channel - number of input feature maps
    Height - input height dimension
    Width - input width dimension
    K - kernel height and width (K x K)
    */
    const int Width_out = (Width - K) + 1;
    const int Height_out = Height - K + 1;
    int b = blockIdx.z;
    int by = blockIdx.y, bx = blockIdx.x, ty = threadIdx.y, tx = threadIdx.x;
    int row = by * TILE_WIDTH + ty, col = bx * TILE_WIDTH + tx;
    int unrolled = Channel * K * K;
    int hw = Height_out * Width_out;
    __shared__ half tileMask[TILE_WIDTH][TILE_WIDTH];
    __shared__ half tileInput[TILE_WIDTH][TILE_WIDTH];
    __shared__ float tileOutput[TILE_WIDTH][TILE_WIDTH];
    wmma::fragment<wmma::matrix_a, TILE_WIDTH, TILE_WIDTH, TILE_WIDTH, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, TILE_WIDTH, TILE_WIDTH, TILE_WIDTH, half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, TILE_WIDTH, TILE_WIDTH, TILE_WIDTH, float> c_frag;
    wmma::fill_fragment(c_frag, 0.0f);
    #define in_4d(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
    int siu = ceil((float) unrolled / TILE_WIDTH);
    int ur = siu / NUM_UNROLL;
    for (int i = 0; i < ur; i++) {
        #pragma unroll
        for (int j = 0; j < NUM_UNROLL; j++) {
            int k = i * NUM_UNROLL + j;
            int ce = k * TILE_WIDTH + tx;
            if (row < Map_out && ce < unrolled) {
                tileMask[ty][tx] = __float2half(mask[row * unrolled + ce]);
            } else {
                tileMask[ty][tx] = __float2half(0);
            }
            int uv = k * TILE_WIDTH + ty;
            if (uv < unrolled && col < hw) {
                int c = uv / (K * K);
                int rem = uv % (K * K);
                int p = rem / K;
                int q = rem % K;
                int h = col / Width_out + p;
                int w = col % Width_out + q;
                tileInput[ty][tx] = __float2half(in_4d(b, c, h, w));
            } else {
                tileInput[ty][tx] = __float2half(0);
            }
            __syncthreads();
            if (ty < 2) {
                wmma::load_matrix_sync(a_frag, (half*) tileMask, TILE_WIDTH);
                wmma::load_matrix_sync(b_frag, (half*) tileInput, TILE_WIDTH);
                wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
            }
            __syncthreads();
        }
    }
    for (int i = NUM_UNROLL * ur; i < siu; i++) {
        int uv = i * TILE_WIDTH + ty;
        if (uv < unrolled && col < hw) {
            int c = uv / (K * K);
            int rem = uv % (K * K);
            int p = rem / K;
            int q = rem % K;
            int h = col / Width_out + p;
            int w = col % Width_out + q;
            tileInput[ty][tx] = __float2half(in_4d(b, c, h, w));
        } else {
            tileInput[ty][tx] = __float2half(0);
        }
        int j = i * TILE_WIDTH + tx;
        if (row < Map_out && j < unrolled) {
            tileMask[ty][tx] = __float2half(mask[row * unrolled + j]);
        } else {
            tileMask[ty][tx] = __float2half(0);
        }
        __syncthreads();
        if (ty < 2) {
            wmma::load_matrix_sync(a_frag, (half*) tileMask, TILE_WIDTH);
            wmma::load_matrix_sync(b_frag, (half*) tileInput, TILE_WIDTH);
            wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
        }
        __syncthreads();
    }
    if (ty < 2) {
        wmma::store_matrix_sync((float*) tileOutput, c_frag, TILE_WIDTH, wmma::mem_row_major);
    }
    __syncthreads();
    if (row < Map_out && col < hw) {
        output[b * Map_out * hw + row * hw + col] = tileOutput[ty][tx];
    }
    #undef in_4d
}
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // TODO: Allocate memory and copy over the relevant data structures to the GPU
    hipMalloc((void **) device_input_ptr, Batch * Channel * Height * Width * sizeof(float));
    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.
    hipMalloc((void **) device_output_ptr, Batch * Map_out * (Height - K + 1) * (Width - K + 1) * sizeof(float));
    hipMalloc((void **) device_mask_ptr, Map_out * Channel * K * K * sizeof(float));
    hipMemcpy(*device_input_ptr, host_input, Batch * Channel * Height * Width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*device_mask_ptr, host_mask, Map_out * Channel * K * K * sizeof(float), hipMemcpyHostToDevice);
    // Useful snippet for error checking
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
        exit(-1);
    }
}
__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // TODO: Set the kernel dimensions and call the fused kernel
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    dim3 DimGrid(ceil((float) Height_out * Width_out / TILE_WIDTH), ceil((float) Map_out / TILE_WIDTH), Batch);
    dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    matmul_conv_fused<<<DimGrid, DimBlock>>>(device_mask, device_input, device_output, Batch, Map_out, Channel, Height, Width, K);
}
__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // TODO: Copy the output back to host
    hipMemcpy(host_output, device_output, Batch * Map_out * (Height - K + 1) * (Width - K + 1) * sizeof(float), hipMemcpyDeviceToHost);
    // TODO: Free device memory
    hipFree(device_input);
    hipFree(device_output);
    hipFree(device_mask);
}
__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}