#include "hip/hip_runtime.h"
#include "matmul.h"

// Tiled matrix multiplication kernel. Computes C = AB
// You don't need to modify this kernel.
__global__ void matrixMultiplyShared(const float *A, const float *B, float *C,
                                     int numARows, int numAColumns,
                                     int numBRows, int numBColumns,
                                     int numCRows, int numCColumns)
{
    __shared__ float tileA[MATMUL_TILE_WIDTH][MATMUL_TILE_WIDTH];
    __shared__ float tileB[MATMUL_TILE_WIDTH][MATMUL_TILE_WIDTH];

    int by = blockIdx.y, bx = blockIdx.x, ty = threadIdx.y, tx = threadIdx.x;

    int row = by * MATMUL_TILE_WIDTH + ty, col = bx * MATMUL_TILE_WIDTH + tx;
    float val = 0;

    for (int tileId = 0; tileId < (numAColumns - 1) / MATMUL_TILE_WIDTH + 1; tileId++) {
        if (row < numARows && tileId * MATMUL_TILE_WIDTH + tx < numAColumns) {
            tileA[ty][tx] = A[(size_t) row * numAColumns + tileId * MATMUL_TILE_WIDTH + tx];
        } else {
            tileA[ty][tx] = 0;
        }
        if (col < numBColumns && tileId * MATMUL_TILE_WIDTH + ty < numBRows) {
            tileB[ty][tx] = B[((size_t) tileId * MATMUL_TILE_WIDTH + ty) * numBColumns + col];
        } else {
            tileB[ty][tx] = 0;
        }
        __syncthreads();

        if (row < numCRows && col < numCColumns) {
            for (int i = 0; i < MATMUL_TILE_WIDTH; i++) {
                val += tileA[ty][i] * tileB[i][tx];
            }
        }
        __syncthreads();
    }

    if (row < numCRows && col < numCColumns) {
        C[row * numCColumns + col] = val;
    }
}