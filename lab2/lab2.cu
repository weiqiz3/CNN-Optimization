#include "hip/hip_runtime.h"
#include <wb.h>
#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)
// Compute C = A * B
__global__ void matrixMultiply(float *A, float *B, float *C, int numARows,
                               int numAColumns, int numBRows,
                               int numBColumns, int numCRows,
                               int numCColumns)
{
  //@@ Implement matrix multiplication kernel here
  // Calculate the row index of the d_P element and d_M
  int Row = blockIdx.y * blockDim.y + threadIdx.y;
  // Calculate the column idenx of d_P and d_N
  int Col = blockIdx.x * blockDim.x + threadIdx.x;
  if ((Row < numCRows) && (Col < numCColumns)) {
    float p = 0;
    // each thread computes one element of the block sub-matrix
    for (int k = 0; k < numAColumns; k++) {
      p += A[Row * numAColumns + k] * B[k * numBColumns + Col];
    }
    C[Row * numCColumns + Col] = p;
  }
}
int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)
  args = wbArg_read(argc, argv);
  //@@ Importing data and creating memory on host
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);
  //@@ Set numCRows and numCColumns
  numCRows = numARows;
  numCColumns = numBColumns;
  //@@ Allocate the hostC matrix
  hostC = (float *) malloc(numCColumns * numCRows * sizeof(float));
  wbLog(TRACE, "The dimensions of C are ", numCRows, " x ", numCColumns);
  //@@ Allocate GPU memory here
  float *devA, *devB, *devC;
  hipMalloc((void **) &devA, numAColumns * numARows * sizeof(float));
  hipMalloc((void **) &devB, numBColumns * numBRows * sizeof(float));
  hipMalloc((void **) &devC, numCColumns * numCRows * sizeof(float));
  //@@ Copy memory to the GPU here
  hipMemcpy(devA, hostA, numAColumns * numARows * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(devB, hostB, numBColumns * numBRows * sizeof(float), hipMemcpyHostToDevice);
  //@@ Initialize the grid and block dimensions here
  dim3 DimGrid(ceil((float) numCColumns / 16), ceil((float) numCRows / 16), 1);
  dim3 DimBlock(16, 16, 1);
  //@@ Launch the GPU Kernel here
  matrixMultiply<<<DimGrid, DimBlock>>>(devA, devB, devC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);
  hipDeviceSynchronize();
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostC, devC, numCColumns * numCRows * sizeof(float), hipMemcpyDeviceToHost);
  //@@ Free the GPU memory here
  hipFree(devA);
  hipFree(devB);
  hipFree(devC);
  wbSolution(args, hostC, numCRows, numCColumns);
  free(hostA);
  free(hostB);
  //@@Free the hostC matrix
  free(hostC);
  return 0;
}