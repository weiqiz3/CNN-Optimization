#include "hip/hip_runtime.h"
#include <wb.h>
#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)
#define TILE_WIDTH 16
// Compute C = A * B
__global__ void matrixMultiplyShared(float *A, float *B, float *C,
                                     int numARows, int numAColumns,
                                     int numBRows, int numBColumns,
                                     int numCRows, int numCColumns) {
  //@@ Insert code to implement matrix multiplication here
  //@@ You have to use shared memory for this MP
  __shared__ float subTileM[TILE_WIDTH][TILE_WIDTH];
  __shared__ float subTileN[TILE_WIDTH][TILE_WIDTH];
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int Row = by * TILE_WIDTH + ty;
  int Col = bx * TILE_WIDTH + tx;
  float p = 0;
  for (int q = 0; q < (numAColumns - 1) / TILE_WIDTH + 1; q++) {
    if (Row < numARows && q * TILE_WIDTH + tx < numAColumns) {
      subTileM[ty][tx] = A[Row * numAColumns + q * TILE_WIDTH + tx];
    } else {
      subTileM[ty][tx] = 0;
    }
    if (Col < numBColumns && q * TILE_WIDTH + ty < numBRows) {
      subTileN[ty][tx] = B[(q * TILE_WIDTH + ty) * numBColumns + Col];
    } else {
      subTileN[ty][tx] = 0;
    }
    __syncthreads();
    if (Row < numCRows && Col < numCColumns) {
      for (int k = 0; k < TILE_WIDTH; k++) {
        p += subTileM[ty][k] * subTileN[k][tx];
      }
    }
    __syncthreads();
  }
  if (Row < numCRows && Col < numCColumns) {
    C[Row * numCColumns + Col] = p;
  }
}
int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)
  args = wbArg_read(argc, argv);
  //@@ Importing data and creating memory on host
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
  //@@ Set numCRows and numCColumns
  numCRows = numARows;
  numCColumns = numBColumns;
  //@@ Allocate the hostC matrix
  hostC = (float *) malloc(numCColumns * numCRows * sizeof(float));
  wbLog(TRACE, "The dimensions of C are ", numCRows, " x ", numCColumns);
  //@@ Allocate GPU memory here
  float *devA, *devB, *devC;
  hipMalloc((void **) &devA, numAColumns * numARows * sizeof(float));
  hipMalloc((void **) &devB, numBColumns * numBRows * sizeof(float));
  hipMalloc((void **) &devC, numCColumns * numCRows * sizeof(float));
  //@@ Copy memory to the GPU here
  hipMemcpy(devA, hostA, numAColumns * numARows * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(devB, hostB, numBColumns * numBRows * sizeof(float), hipMemcpyHostToDevice);
  //@@ Initialize the grid and block dimensions here
  dim3 DimGrid(ceil((float) numCColumns / 16), ceil((float) numCRows / 16), 1);
  dim3 DimBlock(16, 16, 1);
  //@@ Launch the GPU Kernel here
  matrixMultiplyShared<<<DimGrid, DimBlock>>>(devA, devB, devC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);
  hipDeviceSynchronize();
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostC, devC, numCColumns * numCRows * sizeof(float), hipMemcpyDeviceToHost);
  //@@ Free the GPU memory here
  hipFree(devA);
  hipFree(devB);
  hipFree(devC);
  wbSolution(args, hostC, numCRows, numCColumns);
  free(hostA);
  free(hostB);
  //@@ Free the hostC matrix
  free(hostC);
  return 0;
}